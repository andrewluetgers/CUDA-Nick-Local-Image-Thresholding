#include "hip/hip_runtime.h"
/*
 * 
 * NICK Local Image thresholding Algorithm
 * M. Hassan Najafi
 * Najaf011@umn.edu
 */
 
// includes, system
#include <stdlib.h>
#include <stdio.h> //FILE*, fopen, fclose, fread, fwrite
#include <string.h>
#include <math.h>
#include <time.h> //clock_t, clock

// includes, project
//#include <cutil.h>

// includes, kernels
#include "Nick_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void NickOnDevice(const Matrix N, Matrix P_global, Matrix P_shared1, Matrix P_shared2);
bool comparefe(float* A, float* B, int len, float tol);

float cpu_run_time;
float gpu_run_time;
float gpu_run_time_shared1;
float gpu_run_time_shared2;
float gpu_total_time;
float gpu_total_time_shared1;
float gpu_total_time_shared2;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {


	
	//Matrix  M;
	Matrix  N;
	Matrix  P_global;
	Matrix  P_shared1;
	Matrix  P_shared2;
	
	srand(2012);
       int sizex, sizey;
	
	if(argc != 4 && argc != 3) 
	{
		// Allocate and initialize the matrices
		//M  = AllocateMatrix(WINDOW_SIZE, WINDOW_SIZE, 1);
		N  = AllocateMatrix((rand() % 1024) + 1, (rand() % 1024) + 1, 1);
		P_global  = AllocateMatrix(N.height, N.width, 0);
		P_shared1  = AllocateMatrix(N.height, N.width, 0);
		P_shared2  = AllocateMatrix(N.height, N.width, 0);
	}
	else
	{
		// Allocate and read in matrices from disk
                sizex = atoi(argv[1]);
                sizey = atoi(argv[2]);

		//M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 0);
		N  = AllocateMatrix(sizex, sizey, 0);		
		P_global  = AllocateMatrix(sizex, sizey, 0);
		P_shared1  = AllocateMatrix(sizex, sizey, 0);
		P_shared2  = AllocateMatrix(sizex, sizey, 0);
		//(void)ReadFile(&M, argv[2]);
		(void)ReadFile(&N, argv[2]);
	}

	printf("=====================================================\n" );
	printf("Input Matrix Dimension = %d x %d\n", N.height,N.width  );

    NickOnDevice(N, P_global, P_shared1, P_shared2);
    
    // compute the matrix convolution on the CPU for comparison
    Matrix reference = AllocateMatrix(P_global.height, P_global.width, 0);
	
	clock_t start = clock();
        float cpu_run_time;
    computeGold(reference.elements, N.elements, N.height, N.width);
        cpu_run_time = (clock()-start)*1000./CLOCKS_PER_SEC;
	
	printf("Elapsed time for running CPU kernel = %f ms\n", cpu_run_time );
	
	printf("\nSpeedup CPU/GPU Kernel (global)= %f \n", cpu_run_time/gpu_run_time );
	printf("Speedup CPU/GPU Kernel(shared1)= %f \n", cpu_run_time/gpu_run_time_shared1 );
	printf("Speedup CPU/GPU Kernel(shared2)= %f \n\n", cpu_run_time/gpu_run_time_shared2 );
	
	printf("Speedup CPU/GPU Total (global)= %f \n", cpu_run_time/gpu_total_time );
	printf("Speedup CPU/GPU Total (shared1)= %f \n", cpu_run_time/gpu_total_time_shared1 );
	printf("Speedup CPU/GPU Total (shared2)= %f \n\n", cpu_run_time/gpu_total_time_shared2 );
	
    // in this case check if the result is equivalent to the expected soluion
    bool res_global = comparefe(reference.elements, P_global.elements, P_global.width * P_global.height, 0.001f);
	bool res_shared1 = comparefe(reference.elements, P_shared1.elements, P_shared1.width * P_shared1.height, 0.001f);
	bool res_shared2 = comparefe(reference.elements, P_shared2.elements, P_shared2.width * P_shared2.height, 0.001f);
    printf("Test CUDA kernel global %s \n", (1 == res_global) ? "PASSED" : "FAILED");
	printf("Test CUDA kernel shared1 %s \n", (1 == res_shared1) ? "PASSED" : "FAILED");
	printf("Test CUDA kernel shared2 %s \n", (1 == res_shared2) ? "PASSED" : "FAILED");
	
	
//	float p_out;
//	float r_out;
//	int Miss_Match_count=0;
//	  for(int row = 0; row < P.height; ++row) {
//	      for(int col = 0; col < P.width; ++col) {	  
//			  p_out = P.elements[row*P.width + col];
//			  r_out = reference.elements[row*P.width + col];
//			  if (p_out-r_out>0.001f | r_out-p_out>0.001f)
//			  //	printf("Miss match : P[%d]  Produced : %.5f, Expected : %.5f\n ",row*P.width + col, p_out, r_out );
//				Miss_Match_count++;
//		  }
//	  }
//		printf("Total Number of Miss matches: %d\n", Miss_Match_count);
//		printf("=====================================================\n" );
	
    
    if(argc == 4)
    {
		WriteFile(P_global, argv[3]);
	}
 

    FreeMatrix(&N);
    FreeMatrix(&P_global);
	FreeMatrix(&P_shared1);
	FreeMatrix(&P_shared2);
	return 0;
}



////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void NickOnDevice(const Matrix N, Matrix P_global, Matrix P_shared1, Matrix P_shared2)
{
//	float Overhead_allocation;
	float Overhead_Copy;
	//Define a timer from cutil.h
	//MyTimer_overhead is for measuring overhead time
	
	//MyTimer_kernel is for measuring  time elapsed for running kernels
 
	//Allocating memory and measuring their overhead time
        clock_t start = clock();
	Matrix Nd = AllocateDeviceMatrix(N);
	Matrix Pd_global = AllocateDeviceMatrix(P_global);	
	Matrix Pd_shared1 = AllocateDeviceMatrix(P_shared1);
	Matrix Pd_shared2 = AllocateDeviceMatrix(P_shared2);
        Overhead_Copy = (clock() - start) * 1000./CLOCKS_PER_SEC;	
//	Overhead_allocation = cutGetTimerValue(MyTimer_overhead);
//	printf("Elapsed time - Overhead - Allocating = %f ms\n", Overhead_allocation );
	
        start = clock();
	//Copy N to constant Memory
	//hipMemcpyToSymbol(HIP_SYMBOL(Nc), N.elements , IMAGE_WIDTH*IMAGE_HEIGTH*sizeof(float));
	//Copy N to Device Memory 
    CopyToDeviceMatrix(Nd, N);	    
    //Clearing memory
    CopyToDeviceMatrix(Pd_global, P_global);	
        Overhead_Copy += (clock() - start) * 1000./CLOCKS_PER_SEC;	
	
	
	CopyToDeviceMatrix(Pd_shared1, P_shared1);
	CopyToDeviceMatrix(Pd_shared2, P_shared2);	
	
	//Sets 16 KB for shared memory and 48 KB for SM's L1 cache	
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	
    // Setup the execution configuration	
	dim3 dimGrid, dimBlock, dimBlock_Shared1, dimBlock_Shared2;

 	dimBlock.x = BLOCK_SIZE ;
 	dimBlock.y = BLOCK_SIZE ;
 	dimBlock.z = 1;	
	
	dimBlock_Shared1.x = BLOCK_SIZE ;
 	dimBlock_Shared1.y = BLOCK_SIZE ;
 	dimBlock_Shared1.z = 1;	
	
	dimBlock_Shared2.x = BLOCK_SIZE+WINDOW_SIZE-1 ;
	dimBlock_Shared2.y = BLOCK_SIZE+WINDOW_SIZE-1 ;
	dimBlock_Shared2.z = 1 ;
	
 	dimGrid.x = (Nd.width+BLOCK_SIZE-1)/BLOCK_SIZE;
	dimGrid.y = (Nd.height+BLOCK_SIZE-1)/BLOCK_SIZE;
	dimGrid.z = 1;
	
	
	//1. All accesses from global memory
        start = clock();
			NickKernel<<<dimGrid, dimBlock>>>( Nd, Pd_global ,Nd.width, Nd.height );
 	hipDeviceSynchronize();	
	gpu_run_time = (clock() - start)*1000./CLOCKS_PER_SEC;
	
	//2. Some accesses from shared memory and some from global memory
        start = clock();
			NickKernel_shared1<<<dimGrid, dimBlock_Shared1>>>( Nd, Pd_shared1, Nd.width, Nd.height );
 	hipDeviceSynchronize();	
	gpu_run_time_shared1 = (clock() - start)*1000./CLOCKS_PER_SEC;
	
	//3. All accesses from shared memory
        start = clock();
		NickKernel_shared2<<<dimGrid, dimBlock_Shared2>>>( Nd, Pd_shared2, Nd.width, Nd.height );
 	hipDeviceSynchronize();	
	gpu_run_time_shared2 = (clock() - start)*1000./CLOCKS_PER_SEC;
	
	
	printf("Elapsed time for running Kernels on GPUs (global)= %f ms\n", gpu_run_time );
	printf("Elapsed time for running Kernels on GPUs (shared1) = %f ms\n", gpu_run_time_shared1 );
	printf("Elapsed time for running Kernels on GPUs (shared2) = %f ms\n", gpu_run_time_shared2 );	
	
	start = clock();
    // Read P from the device
    CopyFromDeviceMatrix(P_global, Pd_global); 
        Overhead_Copy += (clock() - start) * 1000./CLOCKS_PER_SEC;	
	CopyFromDeviceMatrix(P_shared1, Pd_shared1);
	CopyFromDeviceMatrix(P_shared2, Pd_shared2);
	
	printf("Elapsed time - Overhead - Copy = %f ms\n", Overhead_Copy );
	


	gpu_total_time = gpu_run_time + Overhead_Copy;
	gpu_total_time_shared1 = gpu_run_time_shared1 + Overhead_Copy;
	gpu_total_time_shared2 = gpu_run_time_shared2 + Overhead_Copy;
	printf("Total GPU time (global)= %f ms\n", gpu_total_time );
	printf("Total GPU time (shared1)= %f ms\n", gpu_total_time_shared1 );
	printf("Total GPU time (shared2)= %f ms\n", gpu_total_time_shared2 );
	
    // Free device matrices
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd_global);
	FreeDeviceMatrix(&Pd_shared1);
	FreeDeviceMatrix(&Pd_shared2);


}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}


Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
		if(rand() % 2)
			M.elements[i] = - M.elements[i];
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

// Read a 16x16 floating point matrix in from file
int ReadFile(Matrix* M, char* file_name)
{
	unsigned int data_read = M->height * M->width;
        FILE* infile=fopen(file_name, "r");
	fread(M->elements, sizeof(float), data_read, infile);
        fclose(infile);
	return data_read;
}

// Write a 16x16 floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
    FILE* outfile=fopen(file_name, "w");
    fread(M.elements, sizeof(float), M.width*M.height, outfile);
    //cutWriteFilef(file_name, M.elements, M.width*M.height,
    //                   0.0001f);
}
bool comparefe(float* A, float* B, int len, float tol)
{
    for (int z=0;z<len;z++)
       if (fabs(A[z]-B[z]) > tol) return false;
    return true;
}
